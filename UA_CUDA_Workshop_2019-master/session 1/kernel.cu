
#include "hip/hip_runtime.h"


#include <iostream>

__global__ // Decoradores (igual que en ts). Por defecto, cualquier cosa no decorada, se delcara como __host__.
void suma_vectores_gpu(float* pA, float* pB, float* pC, const int cN) {
	int idX_ = blockIdx.x * blockDim.x + threadIdx.x ; // La formula para atacar memorias = blockIdx.x * blockDim.x + threadIdx.x (la x es para porque es unidimensional (viene de un struct interno)).
	pC[idX_] = pA[idX_] + pB[idX_];
}

void suma_vectores(float* pA, float* pB, float* pC, const int cN) { // Funcion que suma vectores.
	for (unsigned int i = 0; i < cN; i++) {
		pC[i] = pA[i] + pB[i];
	}
}

int main(void) {
	const int kNumElemets = 25600;
	const size_t kNumBytes = kNumElemets * sizeof(float);

	// PASO 0: Seleccionar el dispositivo (tarjeta grafica).
	hipSetDevice(0); // Tarjeta 0, primera tarjeta que usa el dispositivo.

	// PASO 1: Declaracion de memoria en la GPU.
	float* d_a_ = NULL; // Por convencion se inicializa a NULL.
	float* d_b_ = NULL; // La d_ especifica que la variable se alojará en la GPU (es un convenio en CUDA).
	float* d_c_ = NULL;

	hipMalloc((void **)&d_a_, kNumBytes); // La GPU no entiende de tipos, por ello hay que castear a void, usamos & para pasar el puntero.
	hipMalloc((void **)&d_b_, kNumBytes); // cudaMalloc es igual que malloc pero en la RAM de la GPU (VRAM).
	hipMalloc((void **)&d_c_, kNumBytes); 

	float* h_a_ = (float *)malloc(kNumBytes); // La h_ especifica que la variable se alojará en la CPU (es un convenio en CUDA).
	float* h_b_ = (float *)malloc(kNumBytes); // malloc allocates the requested memory and returns a pointer to it.
	float* h_c_ = (float *)malloc(kNumBytes);

	if (h_a_ == NULL || h_b_ == NULL || h_c_ == NULL) { // Comprobamos que el ordenador tiene memoria suficiente par alojar las variables.
		std::cerr << "Fallo al reservar la memoria \n";
		getchar(); // Para que no se cierre la ventana nada mas mostrar el mensaje de error.
		exit(-1); // Codigo de salida de error predefinido.
	}

	for (unsigned int i = 0; i < kNumElemets; i++) { // Rellenamos los vectores con datos aleatorios.
		h_a_[i] = rand() / (float)RAND_MAX;
		h_b_[i] = rand() / (float)RAND_MAX;
	}

	// PASO 2: Copia de datos CPU a GPU.
	hipMemcpy(d_a_, h_a_, kNumBytes, hipMemcpyHostToDevice); // Mandamos los datos a la VRAM.
	hipMemcpy(d_b_, h_b_, kNumBytes, hipMemcpyHostToDevice); // Destino, Origen, Cantidad de bytes a copiar, flag que especifica en que direccion (de CPU a GPU).

	// PASO 3: Lanzar kernel (ejecutar computo).

	// suma_vectores(h_a_, h_b_, h_c_, kNumElemets);

	const int thread_per_block_ = 256;
	const int blocks_per_grid_ = kNumElemets / thread_per_block_;

	dim3 tpb_(thread_per_block_, 1, 1);
	dim3 bpg_(blocks_per_grid_, 1, 1);

	suma_vectores_gpu<<<bpg_, tpb_>>>(d_a_, d_b_, d_c_, kNumElemets);
	hipError_t error_ = hipGetLastError(); // Si hay un error, se guarda en una porcion de memoria de la GPU, esta funcion te devuelve este valor.

	if (error_ != hipSuccess) {
		std::cerr << "Fallo en el kernel" << hipGetErrorString(error_) << "\n";
		getchar(); // Para que no se cierre la ventana nada mas mostrar el mensaje de error.
		exit(-1); // Codigo de salida de error predefinido.
	}

	hipMemcpy(h_c_, d_c_, kNumBytes, hipMemcpyDeviceToHost); // Nos traemos los datos de vuelta de la VRAM.

	// Comprobamos que los datos son los mismos.
	for (unsigned int i = 0; i < kNumElemets; i++) {
		if (fabs(h_a_[i] + h_b_[i] - h_c_[i]) > 1e-5) { // Si el valor rendondeado tiene un error mayor a un umbral (en este caso 1e-5), se ha calculado mal.
			std::cerr << "Fallo en la posicion: " << i << "\n";
			std::cerr << "Resultado es: " << h_c_[i] << "\n";
			std::cerr << "Se esperaba: " << h_a_[i] + h_b_[i] << "\n";
			getchar();
			exit(-1);
		}
	}

	// PASO 4: Librerar recursos. (destruir).
	free(h_a_);
	free(h_b_);
	free(h_c_);

	hipFree(d_a_);
	hipFree(d_b_);
	hipFree(d_c_);

	hipDeviceReset(); // Elibera y resetea el dispositivo.

	std::cout << "Optimo \n"; // Mensaje de exito.
	getchar();

}
